#include "hip/hip_runtime.h"
/*********************************************************************************************
 * file name  : cuda_global_memory_static.cu
 * author     : justin
 * date       : 2024-3-19
 * brief      : 动态全局内存变量使用示例
***********************************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>
#include "../cuda错误检测/error.cuh"


__global__ void kernel(int *d_x, int d_y[2])
{
    d_y[0] += *d_x;
    d_y[1] += *d_x;

    printf("d_x = %d, d_y[0] = %d, d_y[1] = %d.\n", *d_x, d_y[0], d_y[1]);
}



int main(int argc, char **argv)
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CHECK(hipGetDeviceProperties(&deviceProps, devID));
    std::cout << "运行GPU设备:" << deviceProps.name << std::endl;

    int h_x = 1;
    int h_y[2] = {10, 20};              // 主机内存定义变量

    int *d_x, *d_y;
    CHECK(hipMalloc((int**)&d_x, sizeof(int)));
    CHECK(hipMalloc((int**)&d_y, sizeof(int)*2));
    CHECK(hipMemcpy(d_x, &h_x, sizeof(int), hipMemcpyHostToDevice));      // 注意hipMemcpy的前两个参数均为指针
    CHECK(hipMemcpy(d_y, h_y, sizeof(int)*2, hipMemcpyHostToDevice)); 

    dim3 block(1);
    dim3 grid(1);
    kernel<<<grid, block>>>(d_x, d_y);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_y, d_y, sizeof(int)*2, hipMemcpyDeviceToHost)); 
    printf("h_y[0] = %d, h_y[1] = %d.\n", h_y[0], h_y[1]);

    CHECK(hipDeviceReset());

    return 0;
}