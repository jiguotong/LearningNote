/*********************************************************************************************
 * file name  : cuda_query.cu
 * author     : jiguotong
 * date       : 2024-3-16
 * brief      : 运行时API查询GPU信息
***********************************************************************************************/

#include "../cuda错误检测/error.cuh"
#include <stdio.h>

int main(void)
{
    int device_id = 0;
    CHECK(hipSetDevice(device_id));

    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, device_id));

    printf("Device id:                                 %d\n",
        device_id);
    printf("Device name:                               %s\n",
        prop.name);
    printf("Compute capability:                        %d.%d\n",
        prop.major, prop.minor);
    printf("Amount of global memory:                   %g GB\n",
        prop.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("Amount of constant memory:                 %g KB\n",
        prop.totalConstMem  / 1024.0);
    printf("Amount of L2 chche size:                   %d MB\n",
        prop.l2CacheSize / (1024 * 1024));
    printf("Maximum grid size:                         %d %d %d\n",
        prop.maxGridSize[0], 
        prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Maximum block size:                        %d %d %d\n",
        prop.maxThreadsDim[0], prop.maxThreadsDim[1], 
        prop.maxThreadsDim[2]);
    printf("Number of SMs:                             %d\n",
        prop.multiProcessorCount);
    printf("SM最大驻留线程数量:                         %d \n",
        prop.maxThreadsPerMultiProcessor);
    printf("SM最大驻留线程块数量:                        %d \n",
        prop.maxBlocksPerMultiProcessor);
    printf("Maximum amount of shared memory per block: %g KB\n",
        prop.sharedMemPerBlock / 1024.0);
    printf("Maximum amount of shared memory per SM:    %g KB\n",
        prop.sharedMemPerMultiprocessor / 1024.0);
    printf("Maximum number of registers per block:     %d K\n",
        prop.regsPerBlock / 1024);
    printf("Maximum number of registers per SM:        %d K\n",
        prop.regsPerMultiprocessor / 1024);
    printf("Maximum number of threads per block:       %d\n",
        prop.maxThreadsPerBlock);
    printf("Maximum number of threads per SM:          %d\n",
        prop.maxThreadsPerMultiProcessor);

    return 0;
}